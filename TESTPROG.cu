#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "salloc_new.h"


__global__
void kernel1(Arena<8, int> a)
{
  int tid = threadIdx.x;
  a.chunks[tid].count = tid;   	
 // printf("count for chunk %d = %d\n",tid,a.chunks[tid].count);   	
}


__global__
void kernel2(Arena<8, int> a)
{
  int tid = threadIdx.x;
  printf("count for chunk %d = %d\n",tid,a.chunks[tid].count);   	
}

int main(int argc, char** argv)
{
  Arena<8,int> arena(8);
  kernel1<<<1,8>>>(arena);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    printf("Error: %s\n", hipGetErrorString(err));
  hipDeviceSynchronize();
  printf("kernel1 done.\n");
  kernel2<<<1,8>>>(arena);
   err = hipGetLastError();
  if (err != hipSuccess)
    printf("Error: %s\n", hipGetErrorString(err));

  printf("kernel2 done.\n");
  hipDeviceSynchronize();
  return 0;
}
