#include "hip/hip_runtime.h"
// This is the driver program for SALLOC

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "salloc.h"

#define CHUNK_SZ 50 // size of a chunk
#define CAP 1600 // number of chunks in the chunk

typedef char T1; // 


__global__ void kernel(GPUArena<CHUNK_SZ,T1> a )

{
 a.get_new_chunk();
}

__global__ void kernel1(GPUArena<CHUNK_SZ,T1> a, T1* v1, T1* v2, T1* v3 )
{
  unsigned tid = threadIdx.x;
   a.push_back(v1,tid); printf("push_back in v1\n");
   a.push_back(v2,tid); printf("push_back in v2\n");

}

__global__ void kernel5(GPUArena<CHUNK_SZ,T1> a, T1* v2)
{
  unsigned tid = threadIdx.x;
   a.push_back(v2,tid); printf("push_back in v1\n");

}


__global__ void kernel6(GPUArena<CHUNK_SZ,T1> a, T1* v3)
{
  unsigned tid = threadIdx.x;
   a.push_back(v3,tid); printf("push_back in v1\n");

}

__global__ void kernel2(GPUArena<CHUNK_SZ,T1> a, T1* v1, T1* v3)
{
  unsigned tid = threadIdx.x;
  printf("pop value from v1 = %d\n",a.pop_back(v1)); //printf("pop_back from v1\n");
   a.push_back(v3,tid); printf("push_back in v3\n");
  printf("pop value from v3 = %d\n",a.pop_back(v3)); //printf("pop_back from v1\n");
}


__global__ void kernel3(GPUArena<CHUNK_SZ,T1> a, T1* v1, T1* v2)
{
  unsigned tid = threadIdx.x;
  a.push_back(v2,tid);
}

__global__ void kernel4(GPUArena<CHUNK_SZ,T1> a, T1* v1)
{
  unsigned tid = threadIdx.x;
  printf("global index of v1[%d] = %d\n",tid,a.getIndex(v1,tid));
}

int main(int argc, char** argv)
{
  GPUArena<CHUNK_SZ, T1> arena(CAP);
  
  T1 * v1 = arena.createVector(); 
  T1 * v2 = arena.createVector(); 
  T1 * v3 = arena.createVector(); // we can have a parameter 'size' which can be set to CHUNK_SZ by default.
  kernel1<<<1,23>>>(arena, v1, v2, v3);
  //kernel5<<<1,20>>>(arena, v2);
  //kernel6<<<1,20>>>(arena, v3);
  kernel2<<<1,25>>>(arena, v1,v3);
//  kernel1<<<1,5>>>(arena, v1,v2,v3);
//  kernel2<<<1,10>>>(arena, v1,v2,v3);
//  kernel3<<<1,25>>>(arena, v1);
  kernel4<<<1,10>>>(arena, v1);
  hipDeviceSynchronize();
  return 0;
}
