#include "hip/hip_runtime.h"
// This is the driver program for SALLOC

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "salloc_phase3.h"

#define CHUNK_SZ 32 // size of a chunk
#define CAP 16 // number of chunks in the chunk

typedef int T1; // 


__global__ void kernel(GPUArena<CHUNK_SZ,T1> a )

{
 a.get_new_chunk();
}

__global__ void kernel1(GPUArena<CHUNK_SZ,T1> a, GPUChunk<CHUNK_SZ,T1>* v)
{
  printf("%p\n",v); 
  v->values[0] = 5; 
  v->values[1] = 2; 
  printf("%d\n",v->values[0]); 
  //printf("%d\n",*(v + sizeof(T1))); 
}

int main(int argc, char** argv)
{
  GPUArena<CHUNK_SZ, T1> arena(CAP);
  
/* This is the desired API for create vector  */
// GPUChunk<CHUNK_SZ, T1> * v1 = arena.createVector(); // returns the address of the next fully free chunk in arena(on GPU) and stores it on v1 on the CPU
  kernel<<<1,8>>>(arena); 
  hipDeviceSynchronize();
  GPUChunk<CHUNK_SZ,T1> * v1 = arena.createVector(); // 'v1' points to a chunk and not to the array inside the chunk.
  kernel1<<<1,1>>>(arena, v1);
  hipDeviceSynchronize();
  return 0;
}
